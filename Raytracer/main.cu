#include "hip/hip_runtime.h"
#define GLM_ENABLE_EXPERIMENTAL
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <iostream>
#include <chrono>

#include "obj_loader.h"
#include "utils.h"
#include "Object.h"
#include "camera.h"
#include "scene.h"




// CUDA kernel
__global__ void render_kernel(hipSurfaceObject_t surface, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    uchar4 pixel = make_uchar4(x % 256, y % 256, 128, 255);
    surf2Dwrite(pixel, surface, x * sizeof(uchar4), y);
}

// Globals
GLuint tex;
hipGraphicsResource* cuda_tex_resource;
int width = 1920, height = 1080;

// Setup GL texture
void createTexture() {
    glGenTextures(1, &tex);
    glBindTexture(GL_TEXTURE_2D, tex);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, width, height, 0,
        GL_RGBA, GL_UNSIGNED_BYTE, nullptr);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glBindTexture(GL_TEXTURE_2D, 0);

    // Register with CUDA
    hipGraphicsGLRegisterImage(&cuda_tex_resource, tex,
        GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore);
}

float runCuda() {
    hipArray_t array;
    hipGraphicsMapResources(1, &cuda_tex_resource);
    hipGraphicsSubResourceGetMappedArray(&array, cuda_tex_resource, 0, 0);

    hipResourceDesc desc{};
    desc.resType = hipResourceTypeArray;
    desc.res.array.array = array;

    hipSurfaceObject_t surface = 0;
    hipCreateSurfaceObject(&surface, &desc);

    //main defn 
    Camera c1(8.0f, 35.0f, height, width);
    Scene s1(c1);
    material m;
    m.albedo = glm::vec3(1.0f, 0.0f, 0.0f);
    Object o1("test2.obj", glm::vec3(0.0f, 0.0f, -10.0f), glm::vec3(0.0f, 0.0f, 0.0f), m);
    o1.BuildBVH(0, o1.Tri_index.size());
    s1.addObj(o1);
    s1.gen_global_BB();




    //end def

    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    // CUDA Timing
    hipEvent_t start, stop;
    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    render_kernel << <grid, block >> > (surface, width, height);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipDestroySurfaceObject(surface);
    hipGraphicsUnmapResources(1, &cuda_tex_resource);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return milliseconds;
}

// Simple fullscreen quad (fixed pipeline for demo)
void drawTexture() {
    glEnable(GL_TEXTURE_2D);
    glBindTexture(GL_TEXTURE_2D, tex);
    glBegin(GL_QUADS);
    glTexCoord2f(0, 0); glVertex2f(-1, 1); // bottom-left
    glTexCoord2f(1, 0); glVertex2f(1, 1);  // bottom-right
    glTexCoord2f(1, 1); glVertex2f(1, -1);   // top-right
    glTexCoord2f(0, 1); glVertex2f(-1, -1);  // top-left
    glEnd();
}

int main() {
    if (!glfwInit()) return -1;

    GLFWwindow* window = glfwCreateWindow(width, height, "CUDA-GL Texture", nullptr, nullptr);
    if (!window) return -1;
    glfwMakeContextCurrent(window);

    glewInit();
    createTexture();

    // Timing variables
    using Clock = std::chrono::high_resolution_clock;
    auto lastTime = Clock::now();
    int frameCount = 0;

    while (!glfwWindowShouldClose(window)) {
        auto frameStart = Clock::now();

        float kernelTime = runCuda();

        glClear(GL_COLOR_BUFFER_BIT);
        drawTexture();

        glfwSwapBuffers(window);
        glfwPollEvents();

        frameCount++;
        auto now = Clock::now();
        auto elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(now - lastTime).count();
        if (elapsed >= 1000) {
            std::cout << "FPS: " << frameCount << " | Kernel time: " << kernelTime << " ms" << std::endl;
            frameCount = 0;
            lastTime = now;
        }
    }

    hipGraphicsUnregisterResource(cuda_tex_resource);
    glDeleteTextures(1, &tex);
    glfwDestroyWindow(window);
    glfwTerminate();
    return 0;
}
